/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/algo.h"
#include "astra/cuda/2d/par_fp.h"
#include "astra/cuda/2d/fan_fp.h"
#include "astra/cuda/2d/par_bp.h"
#include "astra/cuda/2d/fan_bp.h"
#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"
#include "astra/cuda/2d/astra.h"

#include <cassert>

namespace astraCUDA {

ReconAlgo::ReconAlgo()
{
	parProjs = 0;
	fanProjs = 0;

	useVolumeMask = false;
	useSinogramMask = false;
	D_maskData = 0;
	D_smaskData = 0;

	D_sinoData = 0;
	D_volumeData = 0;

	useMinConstraint = false;
	useMaxConstraint = false;

	freeGPUMemory = false;
}

ReconAlgo::~ReconAlgo()
{
	reset();
}

void ReconAlgo::reset()
{
	delete[] parProjs;
	delete[] fanProjs;

	if (freeGPUMemory) {
		hipFree(D_maskData);
		hipFree(D_smaskData);
		hipFree(D_sinoData);
		hipFree(D_volumeData);
	}

	parProjs = 0;
	fanProjs = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	D_maskData = 0;
	D_smaskData = 0;

	D_sinoData = 0;
	D_volumeData = 0;
	
	useMinConstraint = false;
	useMaxConstraint = false;

	freeGPUMemory = false;
}

bool ReconAlgo::setGPUIndex(int iGPUIndex)
{
	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	return true;
}

bool ReconAlgo::enableVolumeMask()
{
	useVolumeMask = true;
	return true;
}

bool ReconAlgo::enableSinogramMask()
{
	useSinogramMask = true;
	return true;
}


bool ReconAlgo::setGeometry(const astra::CVolumeGeometry2D* pVolGeom,
                            const astra::CProjectionGeometry2D* pProjGeom)
{
	bool ok;

	ok = convertAstraGeometry_dims(pVolGeom, pProjGeom, dims);

	if (!ok)
		return false;

	delete[] parProjs;
	parProjs = 0;
	delete[] fanProjs;
	fanProjs = 0;

	fProjectorScale = 1.0f;
	ok = convertAstraGeometry(pVolGeom, pProjGeom, parProjs, fanProjs, fProjectorScale);
	if (!ok)
		return false;

	return true;
}

bool ReconAlgo::setSuperSampling(int raysPerDet, int raysPerPixelDim)
{
	if (raysPerDet <= 0 || raysPerPixelDim <= 0)
		return false;

	dims.iRaysPerDet = raysPerDet;
	dims.iRaysPerPixelDim = raysPerPixelDim;

	return true;
}

bool ReconAlgo::setVolumeMask(float* _D_maskData, unsigned int _maskPitch)
{
	assert(useVolumeMask);

	D_maskData = _D_maskData;
	maskPitch = _maskPitch;

	return true;
}

bool ReconAlgo::setSinogramMask(float* _D_smaskData, unsigned int _smaskPitch)
{
	assert(useSinogramMask);

	D_smaskData = _D_smaskData;
	smaskPitch = _smaskPitch;

	return true;
}

bool ReconAlgo::setBuffers(float* _D_volumeData, unsigned int _volumePitch,
                      float* _D_projData, unsigned int _projPitch)
{
	D_volumeData = _D_volumeData;
	volumePitch = _volumePitch;
	D_sinoData = _D_projData;
	sinoPitch = _projPitch;

	return true;
}

bool ReconAlgo::setMinConstraint(float fMin)
{
	fMinConstraint = fMin;
	useMinConstraint = true;
	return true;
}

bool ReconAlgo::setMaxConstraint(float fMax)
{
	fMaxConstraint = fMax;
	useMaxConstraint = true;
	return true;
}



bool ReconAlgo::allocateBuffers()
{
	bool ok;
	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		D_volumeData = 0;
		return false;
	}

	if (useVolumeMask) {
		ok = allocateVolumeData(D_maskData, maskPitch, dims);
		if (!ok) {
			hipFree(D_volumeData);
			hipFree(D_sinoData);
			D_volumeData = 0;
			D_sinoData = 0;
			return false;
		}
	}

	if (useSinogramMask) {
		ok = allocateProjectionData(D_smaskData, smaskPitch, dims);
		if (!ok) {
			hipFree(D_volumeData);
			hipFree(D_sinoData);
			hipFree(D_maskData);
			D_volumeData = 0;
			D_sinoData = 0;
			D_maskData = 0;
			return false;
		}
	}

	freeGPUMemory = true;
	return true;
}

bool ReconAlgo::copyDataToGPU(const float* pfSinogram, unsigned int iSinogramPitch,
                              const float* pfReconstruction, unsigned int iReconstructionPitch,
                              const float* pfVolMask, unsigned int iVolMaskPitch,
                              const float* pfSinoMask, unsigned int iSinoMaskPitch)
{
	if (!pfSinogram)
		return false;
	if (!pfReconstruction)
		return false;

	bool ok = copySinogramToDevice(pfSinogram, iSinogramPitch,
	                               dims,
	                               D_sinoData, sinoPitch);
	if (!ok)
		return false;

	ok = copyVolumeToDevice(pfReconstruction, iReconstructionPitch,
	                        dims,
	                        D_volumeData, volumePitch);
	if (!ok)
		return false;



	if (useVolumeMask) {
		if (!pfVolMask)
			return false;

		ok = copyVolumeToDevice(pfVolMask, iVolMaskPitch,
		                        dims,
		                        D_maskData, maskPitch);
		if (!ok)
			return false;
	}

	if (useSinogramMask) {
		if (!pfSinoMask)
			return false;

		ok = copySinogramToDevice(pfSinoMask, iSinoMaskPitch,
		                          dims,
		                          D_smaskData, smaskPitch);
		if (!ok)
			return false;
	}

	return true;
}

bool ReconAlgo::getReconstruction(float* pfReconstruction,
                                  unsigned int iReconstructionPitch) const
{
	bool ok = copyVolumeFromDevice(pfReconstruction, iReconstructionPitch,
	                               dims,
	                               D_volumeData, volumePitch);
	if (!ok)
		return false;

	return true;
}


bool ReconAlgo::callFP(float* D_volumeData, unsigned int volumePitch,
                       float* D_projData, unsigned int projPitch,
                       float outputScale)
{
	if (parProjs) {
		assert(!fanProjs);
		return FP(D_volumeData, volumePitch, D_projData, projPitch,
		          dims, parProjs, fProjectorScale * outputScale);
	} else {
		assert(fanProjs);
		return FanFP(D_volumeData, volumePitch, D_projData, projPitch,
		             dims, fanProjs, fProjectorScale * outputScale);
	}
}

bool ReconAlgo::callBP(float* D_volumeData, unsigned int volumePitch,
                       float* D_projData, unsigned int projPitch,
                       float outputScale)
{
	if (parProjs) {
		assert(!fanProjs);
		return BP(D_volumeData, volumePitch, D_projData, projPitch,
		          dims, parProjs, fProjectorScale * outputScale);
	} else {
		assert(fanProjs);
		return FanBP(D_volumeData, volumePitch, D_projData, projPitch,
		             dims, fanProjs, fProjectorScale * outputScale);
	}

}



}
