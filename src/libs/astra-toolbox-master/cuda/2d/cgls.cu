#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/cgls.h"
#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA {

CGLS::CGLS() : ReconAlgo()
{
	D_z = 0;
	D_p = 0;
	D_r = 0;
	D_w = 0;

	sliceInitialized = false;
}


CGLS::~CGLS()
{
	reset();
}

void CGLS::reset()
{
	hipFree(D_z);
	hipFree(D_p);
	hipFree(D_r);
	hipFree(D_w);

	D_z = 0;
	D_p = 0;
	D_r = 0;
	D_w = 0;

	ReconAlgo::reset();
}

bool CGLS::init()
{
	// Lifetime of z: within an iteration
	allocateVolumeData(D_z, zPitch, dims);

	// Lifetime of p: full algorithm
	allocateVolumeData(D_p, pPitch, dims);

	// Lifetime of r: full algorithm
	allocateProjectionData(D_r, rPitch, dims);
	
	// Lifetime of w: within an iteration
	allocateProjectionData(D_w, wPitch, dims);

	// TODO: check if allocations succeeded
	return true;
}


bool CGLS::setBuffers(float* _D_volumeData, unsigned int _volumePitch,
                      float* _D_projData, unsigned int _projPitch)
{
	bool ok = ReconAlgo::setBuffers(_D_volumeData, _volumePitch,
	                                _D_projData, _projPitch);

	if (!ok)
		return false;

	sliceInitialized = false;

	return true;
}

bool CGLS::copyDataToGPU(const float* pfSinogram, unsigned int iSinogramPitch,
                         const float* pfReconstruction, unsigned int iReconstructionPitch,
                         const float* pfVolMask, unsigned int iVolMaskPitch,
                         const float* pfSinoMask, unsigned int iSinoMaskPitch)
{
	sliceInitialized = false;

	return ReconAlgo::copyDataToGPU(pfSinogram, iSinogramPitch, pfReconstruction, iReconstructionPitch, pfVolMask, iVolMaskPitch, pfSinoMask, iSinoMaskPitch);
}

bool CGLS::iterate(unsigned int iterations)
{
	if (!sliceInitialized) {

		// copy sinogram
		duplicateProjectionData(D_r, D_sinoData, sinoPitch, dims);

		// r = sino - A*x
		if (useVolumeMask) {
			// Use z as temporary storage here since it is unused
			duplicateVolumeData(D_z, D_volumeData, volumePitch, dims);
			processVol<opMul>(D_z, D_maskData, zPitch, dims);
			callFP(D_z, zPitch, D_r, rPitch, -1.0f);
		} else {
			callFP(D_volumeData, volumePitch, D_r, rPitch, -1.0f);
		}


		// p = A'*r
		zeroVolumeData(D_p, pPitch, dims);
		callBP(D_p, pPitch, D_r, rPitch, 1.0f);
		if (useVolumeMask)
			processVol<opMul>(D_p, D_maskData, pPitch, dims);


		gamma = dotProduct2D(D_p, pPitch, dims.iVolWidth, dims.iVolHeight);

		sliceInitialized = true;
	}


	// iteration
	for (unsigned int iter = 0; iter < iterations && !astra::shouldAbort(); ++iter) {

		// w = A*p
		zeroProjectionData(D_w, wPitch, dims);
		callFP(D_p, pPitch, D_w, wPitch, 1.0f);

		// alpha = gamma / <w,w>
		float ww = dotProduct2D(D_w, wPitch, dims.iProjDets, dims.iProjAngles);
		float alpha = gamma / ww;

		// x += alpha*p
		processVol<opAddScaled>(D_volumeData, D_p, alpha, volumePitch, dims);

		// r -= alpha*w
		processSino<opAddScaled>(D_r, D_w, -alpha, rPitch, dims);


		// z = A'*r
		zeroVolumeData(D_z, zPitch, dims);
		callBP(D_z, zPitch, D_r, rPitch, 1.0f);
		if (useVolumeMask)
			processVol<opMul>(D_z, D_maskData, zPitch, dims);

		float beta = 1.0f / gamma;
		gamma = dotProduct2D(D_z, zPitch, dims.iVolWidth, dims.iVolHeight);
		beta *= gamma;

		// p = z + beta*p
		processVol<opScaleAndAdd>(D_p, D_z, beta, pPitch, dims);

	}

	return true;
}


float CGLS::computeDiffNorm()
{
	// We can use w and z as temporary storage here since they're not
	// used outside of iterations.

	// copy sinogram to w
	duplicateProjectionData(D_w, D_sinoData, sinoPitch, dims);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			duplicateVolumeData(D_z, D_volumeData, volumePitch, dims);
			processVol<opMul>(D_z, D_maskData, zPitch, dims);
			callFP(D_z, zPitch, D_w, wPitch, -1.0f);
	} else {
			callFP(D_volumeData, volumePitch, D_w, wPitch, -1.0f);
	}

	// compute norm of D_w

	float s = dotProduct2D(D_w, wPitch, dims.iProjDets, dims.iProjAngles);

	return sqrt(s);
}


}
