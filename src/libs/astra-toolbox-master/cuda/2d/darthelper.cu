#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/darthelper.h"

#include <cassert>

namespace astraCUDA {

// CUDA function for the selection of ROI
__global__ void devRoiSelect(float* in, float radius, unsigned int pitch, unsigned int width, unsigned int height)
{
	float x = (float)(threadIdx.x + 16*blockIdx.x);
	float y = (float)(threadIdx.y + 16*blockIdx.y);

	float w = (width-1.0f)*0.5f;
	float h = (height-1.0f)*0.5f;

	if ((x-w)*(x-w) + (y-h)*(y-h) > radius * radius * 0.25f) 
	{
		float* d = (float*)in;
		unsigned int o = y*pitch+x; 
		d[o] = 0.0f;
	}
}

void roiSelect(float* out, float radius, unsigned int width, unsigned int height)
{
	float* D_data;

	unsigned int pitch;
	// We abuse dims here...
	SDimensions dims;
	dims.iVolWidth = width;
	dims.iVolHeight = height;

	allocateVolumeData(D_data, pitch, dims);
	copyVolumeToDevice(out, width, dims, D_data, pitch);

	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);
	devRoiSelect<<<gridSize, blockSize>>>(D_data, radius, pitch, width, height);

	copyVolumeFromDevice(out, width, dims, D_data, pitch);

	hipFree(D_data);
}




// CUDA function for the masking of DART with a radius == 1
__global__ void devDartMask(float* mask, const float* in, unsigned int conn, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
		float* d = (float*)in;
		float* m = (float*)mask;

		unsigned int o2 = y*pitch+x; // On this row.
		unsigned int o1 = o2 - pitch; // On previous row.
		unsigned int o3 = o2 + pitch; // On next row.

		if ((conn == 8 && // 8-connected
		        (d[o1 - 1] != d[o2] || d[o1] != d[o2] || d[o1 + 1] != d[o2] || 
		         d[o2 - 1] != d[o2] ||                   d[o2 + 1] != d[o2] ||
				 d[o3 - 1] != d[o2] || d[o3] != d[o2] || d[o3 + 1] != d[o2] ))
			|| 
			(conn == 4 && // 4-connected
		        (                      d[o1] != d[o2] ||                      
		         d[o2 - 1] != d[o2] ||                  d[o3 + 1] != d[o2] ||
				                       d[o3] != d[o2]                      )))
		{
			m[o2] = 1.0f;
		}
	}
}


// CUDA function for the masking of DART with a radius > 1
__global__ void devDartMaskRadius(float* mask, const float* in, unsigned int conn, unsigned int radius, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > radius-1 && x < width - radius && y > radius-1 && y < height - radius) 
	{
		float* d = (float*)in;
		float* m = (float*)mask;

		int r = radius;

		// o2: index of the current center pixel
		int o2 = y*pitch+x;

		if (conn == 8) // 8-connected
		{
			for (int row = -r; row <= r; row++) 
			{
				int o1 = (y+row)*pitch+x; 
				for (int col = -r; col <= r; col++) 
				{
					if (d[o1 + col] != d[o2]) {m[o2] = 1.0f; return;}
				}
			}
		}
		else if (conn == 4) // 4-connected
		{
			// horizontal
			unsigned int o1 = y*pitch+x; 
			for (int col = -r; col <= r; col++) 
			{
				if (d[o1 + col] != d[o2]) {m[o2] = 1.0f; return;}
			}

			// vertical
			for (int row = -r; row <= r; row++) 
			{
				unsigned int o1 = (y+row)*pitch+x; 
				if (d[o1] != d[o2]) {m[o2] = 1.0f; return;}
			}
		}
	}
}


// CUDA function for the masking of ADART with a radius == 1
__global__ void devADartMask(float* mask, const float* in, unsigned int conn, unsigned int threshold, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
		float* d = (float*)in;
		float* m = (float*)mask;

		unsigned int o2 = y*pitch+x; // On this row.
		unsigned int o1 = o2 - pitch; // On previous row.
		unsigned int o3 = o2 + pitch; // On next row.

		if (conn == 8)
		{
			if (d[o1 - 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o1    ] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o1 + 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		    if (d[o2 - 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o2 + 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		    if (d[o3 - 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o3    ] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o3 + 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		}
		else if (conn == 4)
		{
			if (d[o1    ] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		    if (d[o2 - 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o2 + 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o3    ] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		}
	}
}


// CUDA function for the masking of ADART with a radius > 1
__global__ void devADartMaskRadius(float* mask, const float* in, unsigned int conn, unsigned int radius, unsigned int threshold, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > radius-1 && x < width - radius && y > radius-1 && y < height - radius)
	{
		float* d = (float*)in;
		float* m = (float*)mask;
	
		int r = radius;

		unsigned int o2 = y*pitch+x; // On this row.

		if (conn == 8)
		{
			for (int row = -r; row <= r; row++) 
			{
				unsigned int o1 = (y+row)*pitch+x; 
				for (int col = -r; col <= r; col++) 
				{
					if (d[o1+col] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
				}
			}
		}
		else if (conn == 4)
		{
			// horizontal
			for (int col = -r; col <= r; col++) 
			{
				if (d[o2+col] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			}

			// vertical
			for (int row = -r; row <= r; row++) 
			{
				unsigned int o1 = (y+row)*pitch+x; 
				if (d[o1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			}
		}
	}
}


void dartMask(float* mask, const float* segmentation, unsigned int conn, unsigned int radius, unsigned int threshold, unsigned int width, unsigned int height)
{
	float* D_segmentationData;
	float* D_maskData;

	unsigned int pitch;
	// We abuse dims here...
	SDimensions dims;
	dims.iVolWidth = width;
	dims.iVolHeight = height;

	allocateVolumeData(D_segmentationData, pitch, dims);
	copyVolumeToDevice(segmentation, width, dims, D_segmentationData, pitch);

	allocateVolumeData(D_maskData, pitch, dims);
	zeroVolumeData(D_maskData, pitch, dims);

	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	if (threshold == 1 && radius == 1)
		devDartMask<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, pitch, width, height);
	else if (threshold > 1 && radius == 1)
		devADartMask<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, threshold, pitch, width, height);
	else if (threshold == 1 && radius > 1)
		devDartMaskRadius<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, radius, pitch, width, height);
	else 
		devADartMaskRadius<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, radius, threshold, pitch, width, height);

	copyVolumeFromDevice(mask, width, dims, D_maskData, pitch);

	hipFree(D_segmentationData);
	hipFree(D_maskData);

}


__global__ void devDartSmoothingRadius(float* out, const float* in, float b, unsigned int radius, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation.
	if (x > radius-1 && x < width - radius && y > radius-1 && y < height - radius)
	{
		float* d = (float*)in;
		float* m = (float*)out;

		unsigned int o2 = y*pitch+x;
		int r = radius;
                float count = 4*r*(r+1);
		float res = -d[o2];

		for (int row = -r; row <= r; row++) 
		{
			unsigned int o1 = (y+row)*pitch+x; 
			for (int col = -r; col <= r; col++) 
			{
				res += d[o1+col];
			}
		}

		res *= b / count;
		res += (1.0f-b) * d[o2];

		m[o2] = res;
	}
}


__global__ void devDartSmoothing(float* out, const float* in, float b, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
		float* d = (float*)in;
		float* m = (float*)out;

		unsigned int o2 = y*pitch+x; // On this row.
		unsigned int o1 = o2 - pitch; // On previous row.
		unsigned int o3 = o2 + pitch; // On next row.

		m[o2] = (1.0f-b) * d[o2] + b * 0.125f * (d[o1 - 1] + d[o1] + d[o1 + 1] + d[o2 - 1] + d[o2 + 1] + d[o3 - 1] + d[o3] + d[o3 + 1]);
	}
}


void dartSmoothing(float* out, const float* in, float b, unsigned int radius, unsigned int width, unsigned int height)
{
	float* D_inData;
	float* D_outData;

	unsigned int pitch;
	// We abuse dims here...
	SDimensions dims;
	dims.iVolWidth = width;
	dims.iVolHeight = height;

	allocateVolumeData(D_inData, pitch, dims);
	copyVolumeToDevice(in, width, dims, D_inData, pitch);

	allocateVolumeData(D_outData, pitch, dims);
	zeroVolumeData(D_outData, pitch, dims);

	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);
	if (radius == 1)
		devDartSmoothing<<<gridSize, blockSize>>>(D_outData, D_inData, b, pitch, width, height);
	else
		devDartSmoothingRadius<<<gridSize, blockSize>>>(D_outData, D_inData, b, radius, pitch, width, height);

	copyVolumeFromDevice(out, width, dims, D_outData, pitch);

	hipFree(D_outData);
	hipFree(D_inData);

}


}
