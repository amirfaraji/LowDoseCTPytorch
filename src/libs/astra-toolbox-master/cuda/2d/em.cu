#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/2d/em.h"
#include "astra/cuda/2d/util.h"
#include "astra/cuda/2d/arith.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA {


// TODO: ensure non-negativity somewhere??


EM::EM()
{
	D_projData = 0;
	D_tmpData = 0;
	D_pixelWeight = 0;

}


EM::~EM()
{
	reset();
}

void EM::reset()
{
	hipFree(D_projData);
	hipFree(D_tmpData);
	hipFree(D_pixelWeight);

	D_projData = 0;
	D_tmpData = 0;
	D_pixelWeight = 0;

	ReconAlgo::reset();
}


bool EM::init()
{
	allocateVolumeData(D_pixelWeight, pixelPitch, dims);
	zeroVolumeData(D_pixelWeight, pixelPitch, dims);

	allocateVolumeData(D_tmpData, tmpPitch, dims);
	zeroVolumeData(D_tmpData, tmpPitch, dims);

	allocateProjectionData(D_projData, projPitch, dims);
	zeroProjectionData(D_projData, projPitch, dims);

	// We can't precompute pixelWeights when using a volume mask
#if 0 
	if (!useVolumeMask)
#endif
		precomputeWeights();

	// TODO: check if allocations succeeded
	return true;
}

bool EM::precomputeWeights()
{
	zeroVolumeData(D_pixelWeight, pixelPitch, dims);
#if 0
	if (useSinogramMask) {
		callBP(D_pixelWeight, pixelPitch, D_smaskData, smaskPitch);
	} else
#endif
	{
		processSino<opSet>(D_projData, 1.0f, projPitch, dims);
		callBP(D_pixelWeight, pixelPitch, D_projData, projPitch, 1.0f);
	}
	processVol<opInvert>(D_pixelWeight, pixelPitch, dims);

#if 0
	if (useVolumeMask) {
		// scale pixel weights with mask to zero out masked pixels
		processVol<opMul>(D_pixelWeight, D_maskData, pixelPitch, dims);
	}
#endif

	return true;
}

bool EM::iterate(unsigned int iterations)
{
#if 0
	if (useVolumeMask)
		precomputeWeights();
#endif

	// iteration
	for (unsigned int iter = 0; iter < iterations && !astra::shouldAbort(); ++iter) {

		// Do FP of volumeData 
		zeroProjectionData(D_projData, projPitch, dims);
		callFP(D_volumeData, volumePitch, D_projData, projPitch, 1.0f);

		// Divide sinogram by FP (into projData)
		processSino<opDividedBy>(D_projData, D_sinoData, projPitch, dims);

		// Do BP of projData into tmpData
		zeroVolumeData(D_tmpData, tmpPitch, dims);
		callBP(D_tmpData, tmpPitch, D_projData, projPitch, 1.0f);

		// Multiply volumeData with tmpData divided by pixel weights
		processVol<opMul2>(D_volumeData, D_tmpData, D_pixelWeight, pixelPitch, dims);

	}

	return true;
}

float EM::computeDiffNorm()
{
	// copy sinogram to projection data
	duplicateProjectionData(D_projData, D_sinoData, sinoPitch, dims);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			duplicateVolumeData(D_tmpData, D_volumeData, volumePitch, dims);
			processVol<opMul>(D_tmpData, D_maskData, tmpPitch, dims);
			callFP(D_tmpData, tmpPitch, D_projData, projPitch, -1.0f);
	} else {
			callFP(D_volumeData, volumePitch, D_projData, projPitch, -1.0f);
	}


	// compute norm of D_projData

	float s = dotProduct2D(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	return sqrt(s);
}


}
