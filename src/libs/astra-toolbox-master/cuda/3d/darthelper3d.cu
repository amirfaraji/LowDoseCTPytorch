#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/dims3d.h"
#include "astra/cuda/3d/darthelper3d.h"

#include <cassert>

namespace astraCUDA3d {


	// -------------------------------------------------------------------------------------------------------------------------------------------------------------------
	__global__ void devDartSmoothing(hipPitchedPtr out, hipPitchedPtr in, float b, SDimensions3D dims)
	{
		unsigned int x = threadIdx.x + 16*blockIdx.x;
		unsigned int y = threadIdx.y + 16*blockIdx.y;

		// Sacrifice the border pixels to simplify the implementation. 
		if (x > 0 && x < dims.iVolX - 1 && y > 0 && y < dims.iVolY - 1) {
			
			float* d = (float*)in.ptr;
			float* m = (float*)out.ptr;

			unsigned int index;
			unsigned int p = (out.pitch >> 2);

			for (unsigned int z = 0; z <= dims.iVolZ-1; z++) {

				float res = 0.0f;

				// bottom slice
				if (z > 0) {
					index = ((z-1)*dims.iVolY + y) * p + x;
					res += d[index-p-1] + d[index-p] + d[index-p+1] +
						d[index  -1] + d[index  ] + d[index  +1] +
						d[index+p-1] + d[index+p] + d[index+p+1];
				}

				// top slice
				if (z < dims.iVolZ-1) {
					index = ((z+1)*dims.iVolY + y) * p + x;
					res += d[index-p-1] + d[index-p] + d[index-p+1] +
						d[index  -1] + d[index  ] + d[index  +1] +
						d[index+p-1] + d[index+p] + d[index+p+1];
				}
	
				// same slice
				index = (z*dims.iVolY + y) * p + x;
				res += d[index-p-1] + d[index-p] + d[index-p+1] +
					d[index  -1] +              d[index  +1] +
					d[index+p-1] + d[index+p] + d[index+p+1];

				// result
				m[index] = (1.0f-b) * d[index] + b * 0.038461538f * res;

			}

		}
	}

	// -------------------------------------------------------------------------------------------------------------------------------------------------------------------
	void dartSmoothing(float* out, const float* in, float b, unsigned int radius, SDimensions3D dims)
	{
		hipPitchedPtr D_inData;
		D_inData = allocateVolumeData(dims);
		copyVolumeToDevice(in, D_inData, dims);

		hipPitchedPtr D_outData;
		D_outData = allocateVolumeData(dims);
		copyVolumeToDevice(out, D_outData, dims);

		dim3 blockSize(16,16);
		dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+15)/16);

		devDartSmoothing<<<gridSize, blockSize>>>(D_outData, D_inData, b, dims);

		copyVolumeFromDevice(out, D_outData, dims);

		hipFree(D_outData.ptr);
		hipFree(D_inData.ptr);

	}


	// -------------------------------------------------------------------------------------------------------------------------------------------------------------------
	// CUDA function for the masking of DART with a radius == 1
	__global__ void devDartMasking(hipPitchedPtr mask, hipPitchedPtr in, unsigned int conn, SDimensions3D dims)
	{
		unsigned int x = threadIdx.x + 16*blockIdx.x;
		unsigned int y = threadIdx.y + 16*blockIdx.y;

		// Sacrifice the border pixels to simplify the implementation. 
		if (x > 0 && x < dims.iVolX - 1 && y > 0 && y < dims.iVolY - 1) {
			
			float* d = (float*)in.ptr;
			float* m = (float*)mask.ptr;

			unsigned int index;
			unsigned int p = (in.pitch >> 2);

			for (unsigned int z = 0; z <= dims.iVolZ-1; z++) {
				
				unsigned int o2 = (z*dims.iVolY + y) * p + x;
				
				m[o2] = 0.0f;

				// bottom slice
				if (z > 0) {
					index = ((z-1)*dims.iVolY + y) * p + x;
					if ((conn == 26 && 
						(d[index-p-1] != d[o2] || d[index-p] != d[o2] || d[index-p+1] != d[o2] || 
						 d[index  -1] != d[o2] || d[index  ] != d[o2] || d[index  +1] != d[o2] || 
						 d[index+p-1] != d[o2] || d[index+p] != d[o2] || d[index+p+1] != d[o2] ))
						|| 
						(conn == 6 && d[index] != d[o2]))
					{
						m[o2] = 1.0f;
						continue;
					}
				}

				// top slice
				if (z < dims.iVolZ-1) {
					index = ((z+1)*dims.iVolY + y) * p + x;
					if ((conn == 26 && 
						(d[index-p-1] != d[o2] || d[index-p] != d[o2] || d[index-p+1] != d[o2] || 
						 d[index  -1] != d[o2] || d[index  ] != d[o2] || d[index  +1] != d[o2] || 
						 d[index+p-1] != d[o2] || d[index+p] != d[o2] || d[index+p+1] != d[o2] ))
						|| 
						(conn == 6 && d[index] != d[o2]))
					{
						m[o2] = 1.0f;
						continue;
					}
				}

				// other slices
				index = (z*dims.iVolY + y) * p + x;
				if ((conn == 26 && 
					(d[index-p-1] != d[o2] || d[index-p] != d[o2] || d[index-p+1] != d[o2] || 
					 d[index  -1] != d[o2] ||                        d[index  +1] != d[o2] || 
					 d[index+p-1] != d[o2] || d[index+p] != d[o2] || d[index+p+1] != d[o2] ))
					|| 
					(conn == 6 && 
					(                         d[index-p] != d[o2] || 
					 d[index  -1] != d[o2] ||                        d[index  +1] != d[o2] || 
					                          d[index+p] != d[o2]                          )))
				{
					m[o2] = 1.0f;
					continue;
				}

			}

		}
	}


	
	// -------------------------------------------------------------------------------------------------------------------------------------------------------------------
	void dartMasking(float* mask, const float* segmentation, unsigned int conn, unsigned int radius, unsigned int threshold, SDimensions3D dims)
	{
		hipPitchedPtr D_maskData;
		D_maskData = allocateVolumeData(dims);
		copyVolumeToDevice(mask, D_maskData, dims);

		hipPitchedPtr D_segmentationData;
		D_segmentationData = allocateVolumeData(dims);
		copyVolumeToDevice(segmentation, D_segmentationData, dims);

		dim3 blockSize(16,16);
		dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+15)/16);

		if (threshold == 1 && radius == 1)
			devDartMasking<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, dims);
		//else if (threshold > 1 && radius == 1)
		//	devADartMask<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, threshold, pitch, width, height, 1, 1);
		//else if (threshold == 1 && radius > 1)
		//	devDartMaskRadius<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, radius, pitch, width, height, 1, 1);
		//else 
		//	devADartMaskRadius<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, radius, threshold, pitch, width, height, 1, 1);

		copyVolumeFromDevice(mask, D_maskData, dims);

		hipFree(D_maskData.ptr);
		hipFree(D_segmentationData.ptr);

	}
	// -------------------------------------------------------------------------------------------------------------------------------------------------------------------

	bool setGPUIndex(int iGPUIndex)
	{
		if (iGPUIndex != -1) {
			hipSetDevice(iGPUIndex);
			hipError_t err = hipGetLastError();

			// Ignore errors caused by calling hipSetDevice multiple times
			if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
				return false;
		}

		return true;
	}


}
