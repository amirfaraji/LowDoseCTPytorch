#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/sirt3d.h"
#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/arith3d.h"
#include "astra/cuda/3d/cone_fp.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA3d {

SIRT::SIRT() : ReconAlgo3D()
{
	D_maskData.ptr = 0;
	D_smaskData.ptr = 0;

	D_sinoData.ptr = 0;
	D_volumeData.ptr = 0;

	D_projData.ptr = 0;
	D_tmpData.ptr = 0;

	D_lineWeight.ptr = 0;
	D_pixelWeight.ptr = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	useMinConstraint = false;
	useMaxConstraint = false;

	fRelaxation = 1.0f;
}


SIRT::~SIRT()
{
	reset();
}

void SIRT::reset()
{
	hipFree(D_projData.ptr);
	hipFree(D_tmpData.ptr);
	hipFree(D_lineWeight.ptr);
	hipFree(D_pixelWeight.ptr);

	D_maskData.ptr = 0;
	D_smaskData.ptr = 0;

	D_sinoData.ptr = 0;
	D_volumeData.ptr = 0;

	D_projData.ptr = 0;
	D_tmpData.ptr = 0;

	D_lineWeight.ptr = 0;
	D_pixelWeight.ptr = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	fRelaxation = 1.0f;

	ReconAlgo3D::reset();
}

bool SIRT::enableVolumeMask()
{
	useVolumeMask = true;
	return true;
}

bool SIRT::enableSinogramMask()
{
	useSinogramMask = true;
	return true;
}


bool SIRT::init()
{
	D_pixelWeight = allocateVolumeData(dims);
	zeroVolumeData(D_pixelWeight, dims);

	D_tmpData = allocateVolumeData(dims);
	zeroVolumeData(D_tmpData, dims);

	D_projData = allocateProjectionData(dims);
	zeroProjectionData(D_projData, dims);

	D_lineWeight = allocateProjectionData(dims);
	zeroProjectionData(D_lineWeight, dims);

	// We can't precompute lineWeights and pixelWeights when using a mask
	if (!useVolumeMask && !useSinogramMask)
		precomputeWeights();

	// TODO: check if allocations succeeded
	return true;
}

bool SIRT::setMinConstraint(float fMin)
{
	fMinConstraint = fMin;
	useMinConstraint = true;
	return true;
}

bool SIRT::setMaxConstraint(float fMax)
{
	fMaxConstraint = fMax;
	useMaxConstraint = true;
	return true;
}

bool SIRT::precomputeWeights()
{
	zeroProjectionData(D_lineWeight, dims);
	if (useVolumeMask) {
		callFP(D_maskData, D_lineWeight, 1.0f);
	} else {
		processVol3D<opSet>(D_tmpData, 1.0f, dims);
		callFP(D_tmpData, D_lineWeight, 1.0f);
	}
	processSino3D<opInvert>(D_lineWeight, dims);

	if (useSinogramMask) {
		// scale line weights with sinogram mask to zero out masked sinogram pixels
		processSino3D<opMul>(D_lineWeight, D_smaskData, dims);
	}

	zeroVolumeData(D_pixelWeight, dims);

	if (useSinogramMask) {
		callBP(D_pixelWeight, D_smaskData, 1.0f);
	} else {
		processSino3D<opSet>(D_projData, 1.0f, dims);
		callBP(D_pixelWeight, D_projData, 1.0f);
	}
#if 0
	float* bufp = new float[512*512];

	for (int i = 0; i < 180; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)D_projData.ptr)+180*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "ray%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif

#if 0
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)D_pixelWeight.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		char fname[20];
		sprintf(fname, "pix%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif
	processVol3D<opInvert>(D_pixelWeight, dims);

	if (useVolumeMask) {
		// scale pixel weights with mask to zero out masked pixels
		processVol3D<opMul>(D_pixelWeight, D_maskData, dims);
	}
	processVol3D<opMul>(D_pixelWeight, fRelaxation, dims);


	return true;
}


bool SIRT::setVolumeMask(hipPitchedPtr& _D_maskData)
{
	assert(useVolumeMask);

	D_maskData = _D_maskData;

	return true;
}

bool SIRT::setSinogramMask(hipPitchedPtr& _D_smaskData)
{
	assert(useSinogramMask);

	D_smaskData = _D_smaskData;

	return true;
}

bool SIRT::setBuffers(hipPitchedPtr& _D_volumeData,
                      hipPitchedPtr& _D_projData)
{
	D_volumeData = _D_volumeData;
	D_sinoData = _D_projData;

	return true;
}

bool SIRT::iterate(unsigned int iterations)
{
	if (useVolumeMask || useSinogramMask)
		precomputeWeights();

#if 0
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)D_pixelWeight.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		char fname[20];
		sprintf(fname, "pix%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif
#if 0
	float* bufp = new float[512*512];

	for (int i = 0; i < 100; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)D_lineWeight.ptr)+100*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "ray%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif


	// iteration
	for (unsigned int iter = 0; iter < iterations && !astra::shouldAbort(); ++iter) {
		// copy sinogram to projection data
		duplicateProjectionData(D_projData, D_sinoData, dims);

		// do FP, subtracting projection from sinogram
		if (useVolumeMask) {
				duplicateVolumeData(D_tmpData, D_volumeData, dims);
				processVol3D<opMul>(D_tmpData, D_maskData, dims);
				callFP(D_tmpData, D_projData, -1.0f);
		} else {
				callFP(D_volumeData, D_projData, -1.0f);
		}

		processSino3D<opMul>(D_projData, D_lineWeight, dims);

		zeroVolumeData(D_tmpData, dims);
#if 0
	float* bufp = new float[512*512];
	printf("Dumping projData: %p\n", (void*)D_projData.ptr);
	for (int i = 0; i < 180; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)D_projData.ptr)+180*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "diff%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif


		callBP(D_tmpData, D_projData, 1.0f);
#if 0
	printf("Dumping tmpData: %p\n", (void*)D_tmpData.ptr);
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)D_tmpData.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		char fname[20];
		sprintf(fname, "add%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif

		// pixel weights also contain the volume mask and relaxation factor
		processVol3D<opAddMul>(D_volumeData, D_tmpData, D_pixelWeight, dims);

		if (useMinConstraint)
			processVol3D<opClampMin>(D_volumeData, fMinConstraint, dims);
		if (useMaxConstraint)
			processVol3D<opClampMax>(D_volumeData, fMaxConstraint, dims);
	}

	return true;
}

float SIRT::computeDiffNorm()
{
	// copy sinogram to projection data
	duplicateProjectionData(D_projData, D_sinoData, dims);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			duplicateVolumeData(D_tmpData, D_volumeData, dims);
			processVol3D<opMul>(D_tmpData, D_maskData, dims);
			callFP(D_tmpData, D_projData, -1.0f);
	} else {
			callFP(D_volumeData, D_projData, -1.0f);
	}

	float s = dotProduct3D(D_projData, dims.iProjU, dims.iProjAngles, dims.iProjV);
	return sqrt(s);
}


bool doSIRT(hipPitchedPtr& D_volumeData, 
            hipPitchedPtr& D_sinoData,
            hipPitchedPtr& D_maskData,
            const SDimensions3D& dims, const SConeProjection* angles,
            unsigned int iterations)
{
	SIRT sirt;
	bool ok = true;

	ok &= sirt.setConeGeometry(dims, angles, SProjectorParams3D());
	if (D_maskData.ptr)
		ok &= sirt.enableVolumeMask();

	if (!ok)
		return false;

	ok = sirt.init();
	if (!ok)
		return false;

	if (D_maskData.ptr)
		ok &= sirt.setVolumeMask(D_maskData);

	ok &= sirt.setBuffers(D_volumeData, D_sinoData);
	if (!ok)
		return false;

	ok = sirt.iterate(iterations);

	return ok;
}

}

