#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2018, imec Vision Lab, University of Antwerp
           2014-2018, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "astra/cuda/3d/cgls3d.h"
#include "astra/cuda/3d/util3d.h"
#include "astra/cuda/3d/arith3d.h"
#include "astra/cuda/3d/cone_fp.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA3d {

CGLS::CGLS() : ReconAlgo3D()
{
	D_maskData.ptr = 0;
	D_smaskData.ptr = 0;

	D_sinoData.ptr = 0;
	D_volumeData.ptr = 0;

	D_r.ptr = 0;
	D_w.ptr = 0;
	D_z.ptr = 0;
	D_p.ptr = 0;

	useVolumeMask = false;
	useSinogramMask = false;
}


CGLS::~CGLS()
{
	reset();
}

void CGLS::reset()
{
	hipFree(D_r.ptr);
	hipFree(D_w.ptr);
	hipFree(D_z.ptr);
	hipFree(D_p.ptr);

	D_maskData.ptr = 0;
	D_smaskData.ptr = 0;

	D_sinoData.ptr = 0;
	D_volumeData.ptr = 0;

	D_r.ptr = 0;
	D_w.ptr = 0;
	D_z.ptr = 0;
	D_p.ptr = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	sliceInitialized = false;

	ReconAlgo3D::reset();
}

bool CGLS::enableVolumeMask()
{
	useVolumeMask = true;
	return true;
}

bool CGLS::enableSinogramMask()
{
	useSinogramMask = true;
	return true;
}


bool CGLS::init()
{
	D_z = allocateVolumeData(dims);
	D_p = allocateVolumeData(dims);
	D_r = allocateProjectionData(dims);
	D_w = allocateProjectionData(dims);

	// TODO: check if allocations succeeded
	return true;
}

bool CGLS::setVolumeMask(hipPitchedPtr& _D_maskData)
{
	assert(useVolumeMask);

	D_maskData = _D_maskData;

	return true;
}

bool CGLS::setSinogramMask(hipPitchedPtr& _D_smaskData)
{
	return false;
#if 0
	// TODO: Implement this
	assert(useSinogramMask);

	D_smaskData = _D_smaskData;
	return true;
#endif
}

bool CGLS::setBuffers(hipPitchedPtr& _D_volumeData,
                      hipPitchedPtr& _D_projData)
{
	D_volumeData = _D_volumeData;
	D_sinoData = _D_projData;

	sliceInitialized = false;

	return true;
}

bool CGLS::iterate(unsigned int iterations)
{
	if (!sliceInitialized) {

		// copy sinogram
		duplicateProjectionData(D_r, D_sinoData, dims);

		// r = sino - A*x
		if (useVolumeMask) {
				duplicateVolumeData(D_z, D_volumeData, dims);
				processVol3D<opMul>(D_z, D_maskData, dims);
				callFP(D_z, D_r, -1.0f);
		} else {
				callFP(D_volumeData, D_r, -1.0f);
		}

		// p = A'*r
		zeroVolumeData(D_p, dims);
		callBP(D_p, D_r, 1.0f);
		if (useVolumeMask)
			processVol3D<opMul>(D_p, D_maskData, dims);

		gamma = dotProduct3D(D_p, dims.iVolX, dims.iVolY, dims.iVolZ);

		sliceInitialized = true;

	}


	// iteration
	for (unsigned int iter = 0; iter < iterations && !astra::shouldAbort(); ++iter) {

		// w = A*p
		zeroProjectionData(D_w, dims);
		callFP(D_p, D_w, 1.0f);

		// alpha = gamma / <w,w>
		float ww = dotProduct3D(D_w, dims.iProjU, dims.iProjAngles, dims.iProjV);
		float alpha = gamma / ww;

		// x += alpha*p
		processVol3D<opAddScaled>(D_volumeData, D_p, alpha, dims);

		// r -= alpha*w
		processSino3D<opAddScaled>(D_r, D_w, -alpha, dims);

		// z = A'*r
		zeroVolumeData(D_z, dims);
		callBP(D_z, D_r, 1.0f);
		if (useVolumeMask)
			processVol3D<opMul>(D_z, D_maskData, dims);

		float beta = 1.0f / gamma;
		gamma = dotProduct3D(D_z, dims.iVolX, dims.iVolY, dims.iVolZ);

		beta *= gamma;

		// p = z + beta*p
		processVol3D<opScaleAndAdd>(D_p, D_z, beta, dims);
	}

	return true;
}

float CGLS::computeDiffNorm()
{
	// We can use w and z as temporary storage here since they're not
	// used outside of iterations.

	// copy sinogram to w
	duplicateProjectionData(D_w, D_sinoData, dims);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			duplicateVolumeData(D_z, D_volumeData, dims);
			processVol3D<opMul>(D_z, D_maskData, dims);
			callFP(D_z, D_w, -1.0f);
	} else {
			callFP(D_volumeData, D_w, -1.0f);
	}

	float s = dotProduct3D(D_w, dims.iProjU, dims.iProjAngles, dims.iProjV);
	return sqrt(s);
}


bool doCGLS(hipPitchedPtr& D_volumeData, 
            hipPitchedPtr& D_sinoData,
            hipPitchedPtr& D_maskData,
            const SDimensions3D& dims, const SConeProjection* angles,
            unsigned int iterations)
{
	CGLS cgls;
	bool ok = true;

	ok &= cgls.setConeGeometry(dims, angles, SProjectorParams3D());
	if (D_maskData.ptr)
		ok &= cgls.enableVolumeMask();

	if (!ok)
		return false;

	ok = cgls.init();
	if (!ok)
		return false;

	if (D_maskData.ptr)
		ok &= cgls.setVolumeMask(D_maskData);

	ok &= cgls.setBuffers(D_volumeData, D_sinoData);
	if (!ok)
		return false;

	ok = cgls.iterate(iterations);

	return ok;
}

}
